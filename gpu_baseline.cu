#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1024

__global__ void ml_task(float* a, float* b, float* c) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) c[idx] = a[idx] * b[idx];
}

void run_sequential_tasks() {
    float *a, *b, *c;
    hipMallocManaged(&a, N * sizeof(float));
    hipMallocManaged(&b, N * sizeof(float));
    hipMallocManaged(&c, N * sizeof(float));

    for (int i = 0; i < N; i++)
    {
        a[i] = 2.0f; b[i] = 3.0f;
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    for (int i = 0; i < 3; i++)
    {
        ml_task<<<(N + 255)/256, 256>>>(a, b, c);
        hipDeviceSynchronize();
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    printf("Bseline GPU time: %f ms\n", ms);

    hipFree(a); hipFree(b); hipFree(c);
}

int main() {
    run_sequential_tasks();
    return 0;
}
